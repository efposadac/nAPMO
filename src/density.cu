#include "hip/hip_runtime.h"
/*file: density.cu
nAPMO package
Copyright (c) 2015, Edwin Fernando Posada
All rights reserved.
Version: 0.1
efposadac@sissa.it*/

#define THREADS_PER_BLOCK 64

extern "C" {
#include "include/density.h"
}

void density_gto(BasisSet *basis, double *r, double *dens, double *output,
                 int size) {

  int sizeBasis;
  double *dens_d, *x_d, *y_d, *z_d, *output_d;
  BasisSet basis_d;

  /*
  Allocating Space
  */
  basis_set_init(basis, &basis_d);

  /*
  Copy points to device
  */
  hipMalloc((void **)&x_d, size * sizeof(double));
  hipMalloc((void **)&y_d, size * sizeof(double));
  hipMalloc((void **)&z_d, size * sizeof(double));

  double *x = (double *)malloc(size * sizeof(double));
  double *y = (double *)malloc(size * sizeof(double));
  double *z = (double *)malloc(size * sizeof(double));

  for (int i = 0; i < size; ++i) {
    int idx = i * 3;
    x[i] = r[idx + 0];
    y[i] = r[idx + 1];
    z[i] = r[idx + 2];
  }

  hipMemcpy(x_d, x, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, size * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(z_d, z, size * sizeof(double), hipMemcpyHostToDevice);

  free(x);
  free(y);
  free(z);

  /*
  Copy density matrix to device
  */
  sizeBasis = basis->n_cont * basis->n_cont;
  hipMalloc((void **)&dens_d, sizeBasis * sizeof(double));
  hipMemcpy(dens_d, dens, sizeBasis * sizeof(double), hipMemcpyHostToDevice);

  /*
  Allocate space for output
  */
  hipMalloc((void **)&output_d, size * sizeof(double));

  /*
  Calculate the function
  */
  dim3 dimGrid(((size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), 1, 1);

  density_gto_kernel<<<dimGrid, THREADS_PER_BLOCK>>>(basis_d, x_d, y_d, z_d,
                                                     dens_d, output_d, size);
  CUERR

  /*
  Bring result back
  */
  hipMemcpy(output, output_d, size * sizeof(double), hipMemcpyDeviceToHost);

  /*
  clear memory
  */
  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(dens_d);
  hipFree(output_d);
  basis_set_free(&basis_d);
}

__global__ void density_gto_kernel(BasisSet basis, double *x, double *y,
                                   double *z, double *dens, double *output,
                                   int size) {

  const unsigned int point = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  const unsigned int n_cont = basis.n_cont;

  double temp_val, function_value;
  double basis_val[64], r[3];

  if (point < size) {

    r[0] = x[point];
    r[1] = y[point];
    r[2] = z[point];

    basis_set_compute_gto(basis, r, basis_val);

    function_value = 0.0;
    for (int i = 0; i < n_cont; ++i) {

      temp_val = 0.0;
      for (int j = 0; j < n_cont; ++j) {
        temp_val += basis_val[j] * dens[i * n_cont + j];
      }

      function_value += temp_val * basis_val[i];
    }

    output[point] = function_value;
  }
}
